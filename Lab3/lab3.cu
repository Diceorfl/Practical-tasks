
#include <hip/hip_runtime.h>
__global__ void DistKernel(float *a, float *b, int n)
{
 // Определение индекса потока
 int k = threadIdx.x + blockIdx.x*blockDim.x;
 // Обработка соответствующей порции данных
 for(int i = 0; i < n; i++)
 {
   float s = 0.0;
   for(int j = 0; j < n; j++)
     s+= (*(a + k * n + j) - *(a + i * n + j))*(*(a + k * n + j) - *(a + i * n + j));
   *(b + k * n + i) = s;
 }
}

// a, b – указатели на исходные массивы
// n – размер массивов (число элементов)
void dist_cuda(float *a, float *b, int n)
{
 int SizeInBytes = n * n * sizeof(float);
 // Указатели на массивы в видеопамяти
 float *a_gpu = NULL;
 float *b_gpu = NULL;
 // Выделение памяти под массивы на GPU
 hipMalloc( (void **)&a_gpu, SizeInBytes );
 hipMalloc( (void **)&b_gpu, SizeInBytes );
 // Копирование исходных данных из CPU на GPU
 hipMemcpy(a_gpu, a, SizeInBytes, hipMemcpyHostToDevice); // a_gpu = a
 // Задание конфигурации запуска ядра
 dim3 threads = dim3(16, 1); // 16 потоков в блоке
 dim3 blocks = dim3(n/threads.x, 1); // n/16 блоков в сетке
 // Запуск ядра
 DistKernel<<<blocks, threads>>>(a_gpu, b_gpu, n);
 // Копирование результата из GPU в CPU
 hipMemcpy(b, b_gpu, SizeInBytes, hipMemcpyDeviceToHost); // b_gpu = b
 // Освобождение памяти GPU
 hipFree(a_gpu);
 hipFree(b_gpu);
}
